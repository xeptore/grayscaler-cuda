#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <errno.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <memory.h>
#include <jpeglib.h>
#include <pthread.h>
#include "config.h"


const unsigned int NUM_THREADS = BLOCK_THREADS * GRID_BLOCKS;
const unsigned char INPUT_IMAGE_COMPONENTS_NUMBER = 3;
const unsigned char OUTPUT_IMAGE_COMPONENTS_NUMBER = 1;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPU: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ unsigned char calculate_gray(
  const unsigned char red,
  const unsigned char green,
  const unsigned char blue
) {
  return red * 0.2126 + green * 0.7152 + blue * 0.0722;
}

void set_decompressor_options(
  struct jpeg_decompress_struct *decompressor,
  struct jpeg_error_mgr *error_manager,
  FILE *input_file
) {
  decompressor->err = jpeg_std_error(error_manager);
  jpeg_create_decompress(decompressor);
  jpeg_stdio_src(decompressor, input_file);
  (void)jpeg_read_header(decompressor, TRUE);
  (void)jpeg_start_decompress(decompressor);
}

void set_compressor_options(
  struct jpeg_compress_struct *compressor,
  const struct jpeg_decompress_struct *decompressor,
  struct jpeg_error_mgr *error_manager,
  FILE *output_file
) {
  compressor->err = jpeg_std_error(error_manager);
  jpeg_create_compress(compressor);
  jpeg_stdio_dest(compressor, output_file);
  compressor->in_color_space = JCS_GRAYSCALE;
  compressor->input_components = OUTPUT_IMAGE_COMPONENTS_NUMBER;
  jpeg_set_defaults(compressor);
  compressor->image_width = decompressor->output_width;
  compressor->image_height = decompressor->image_height;
  compressor->density_unit = decompressor->density_unit;
  compressor->X_density = decompressor->X_density;
  compressor->Y_density = decompressor->Y_density;
  jpeg_start_compress(compressor, TRUE);
}

size_t calculate_input_image_row_length(
  const struct jpeg_decompress_struct *decompressor
) {
  return decompressor->output_width * decompressor->num_components;
}

struct transform_row_params {
  size_t image_width;
  size_t start_row;
  size_t num_rows;
};

__global__ void transform_rows(
  struct transform_row_params *params,
  unsigned char *input,
  unsigned char *output
) {
  const unsigned long int workerIdx = threadIdx.x + blockDim.x * blockIdx.x;
  const size_t image_width = params[workerIdx].image_width;
  for (size_t i = params[workerIdx].start_row; i < params[workerIdx].start_row + params[workerIdx].num_rows; i++) {
    for (size_t j = 0; j < image_width; j++) {
      const unsigned char red = input[(i * image_width * INPUT_IMAGE_COMPONENTS_NUMBER) + j * INPUT_IMAGE_COMPONENTS_NUMBER + 0];
      const unsigned char green = input[(i * image_width * INPUT_IMAGE_COMPONENTS_NUMBER) + j * INPUT_IMAGE_COMPONENTS_NUMBER + 1];
      const unsigned char blue = input[(i * image_width * INPUT_IMAGE_COMPONENTS_NUMBER) + j * INPUT_IMAGE_COMPONENTS_NUMBER + 2];
      const unsigned char gray = calculate_gray(red, green, blue);
      output[i * image_width + j] = gray;
    }
  }
}

int transform_image(const char *input_filename, const char *output_filename) {
  FILE *input_file = fopen(input_filename, "rb");
  if (!input_file) {
    (void)fprintf(
      stderr,
      "🛑🙁 error opening jpeg file '%s': %s 🙁🛑\n",
      input_filename,
      strerror(errno)
    );
    return errno;
  }

  FILE *output_file = fopen(output_filename, "wb");
  if (!output_file) {
    (void)fprintf(
      stderr,
      "🛑🙁 error opening output jpeg file '%s': %s 🙁🛑\n",
      output_filename,
      strerror(errno)
    );
    return errno;
  }

  struct jpeg_error_mgr error_manager;

  struct jpeg_decompress_struct decompressor;
  set_decompressor_options(&decompressor, &error_manager, input_file);

  if (decompressor.image_height < NUM_THREADS) {
    (void)fprintf(
      stderr,
      "🛑🤔 how is that possible to distribute processing %u rows on %u threads? 🤔🛑\n",
      decompressor.image_height,
      NUM_THREADS
    );
    return 1;

  }

  struct jpeg_compress_struct compressor;
  set_compressor_options(&compressor, &decompressor, &error_manager, output_file);

  const size_t input_image_row_length = calculate_input_image_row_length(&decompressor);
  const unsigned int output_image_row_length = compressor.image_width;

  const size_t IMAGE_HEIGHT = decompressor.image_height;
  const size_t INPUT_IMAGE_SIZE_IN_BYTES = decompressor.image_height * decompressor.image_width * INPUT_IMAGE_COMPONENTS_NUMBER;
  const size_t OUTPUT_IMAGE_SIZE_IN_BYTES = compressor.image_height * compressor.image_width * OUTPUT_IMAGE_COMPONENTS_NUMBER;

  unsigned char *all_buffer = (unsigned char *)malloc(INPUT_IMAGE_SIZE_IN_BYTES + OUTPUT_IMAGE_SIZE_IN_BYTES);
  if (all_buffer == NULL) {
    (void)fprintf(stderr, "failed to allocate enough memory.\n");
    exit(-1);
  }

  unsigned char *input_buffer = &all_buffer[0];

  JSAMPROW scan_rows_buffer[decompressor.image_height];
  for (size_t i = 0; i < decompressor.image_height; i++) {
    scan_rows_buffer[i] = &input_buffer[i * input_image_row_length];
  }

  while (decompressor.output_scanline < decompressor.output_height) {
    (void)jpeg_read_scanlines(
      &decompressor,
      &scan_rows_buffer[decompressor.output_scanline],
      decompressor.output_height - decompressor.output_scanline
    );
  }

  unsigned char *device_input_buffer;
  gpuErrchk(hipMallocManaged(&device_input_buffer, INPUT_IMAGE_SIZE_IN_BYTES));
  const size_t input_image_width_in_bytes = decompressor.image_width * INPUT_IMAGE_COMPONENTS_NUMBER;
  for (size_t i = 0; i < IMAGE_HEIGHT; i++) {
    for (size_t j = 0; j < input_image_width_in_bytes; j++) {
      device_input_buffer[i * input_image_width_in_bytes + j] = scan_rows_buffer[i][j];
    }
  }

  unsigned char *output_buffer = &all_buffer[INPUT_IMAGE_SIZE_IN_BYTES];
  JSAMPROW output_rows_buffer[compressor.image_height];
  for (size_t i = 0; i < compressor.image_height; i++) {
    output_rows_buffer[i] = &output_buffer[i * output_image_row_length];
  }

  struct transform_row_params thread_params_refs[NUM_THREADS];

  const unsigned int quotient = decompressor.image_height / NUM_THREADS;
  const unsigned int remainder = decompressor.image_height % NUM_THREADS;

  unsigned long total_assigned_rows = 0U;
  for (size_t i = 0; i < NUM_THREADS; i++) {
    const unsigned long int worker_quotient = (i < remainder) ? (quotient + 1) : (quotient);
    thread_params_refs[i].image_width = decompressor.image_width;
    thread_params_refs[i].start_row = total_assigned_rows;
    thread_params_refs[i].num_rows = worker_quotient;
    total_assigned_rows += worker_quotient;
  }

  struct transform_row_params *device_param_refs;
  gpuErrchk(hipMallocManaged(&device_param_refs, sizeof(struct transform_row_params) * NUM_THREADS));
  gpuErrchk(hipMemcpy(
    device_param_refs,
    thread_params_refs,
    sizeof(struct transform_row_params) * NUM_THREADS,
    hipMemcpyHostToDevice
  ));

  unsigned char *device_output_buffer;
  gpuErrchk(hipMallocManaged(&device_output_buffer, OUTPUT_IMAGE_SIZE_IN_BYTES));

  dim3 grid_size(GRID_BLOCKS, 1, 1);
  dim3 block_size(BLOCK_THREADS, 1, 1);

  struct timespec start, end;
  timespec_get(&start, TIME_UTC);

  transform_rows<<<grid_size, block_size>>>(device_param_refs, device_input_buffer, device_output_buffer);

  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  timespec_get(&end, TIME_UTC);
  unsigned long int time_in_nano_seconds = (end.tv_sec - start.tv_sec) * 1e9 + (end.tv_nsec - start.tv_nsec);
  printf("%lu\n", time_in_nano_seconds);

  unsigned char *temp = (unsigned char *)malloc(OUTPUT_IMAGE_SIZE_IN_BYTES);
  gpuErrchk(hipMemcpy(
    temp,
    device_output_buffer,
    OUTPUT_IMAGE_SIZE_IN_BYTES,
    hipMemcpyDeviceToHost
  ));
  for (size_t i = 0; i < compressor.image_height; i++) {
    for (size_t j = 0; j < compressor.image_width; j++) {
      output_rows_buffer[i][j] = temp[i * compressor.image_width + j];
    }
  }

  for (size_t i = 0; i < compressor.image_height; i++) {
    (void)jpeg_write_scanlines(&compressor, &output_rows_buffer[i], 1);
  }

  (void)jpeg_finish_decompress(&decompressor);
  jpeg_finish_compress(&compressor);
  jpeg_destroy_decompress(&decompressor);
  jpeg_destroy_compress(&compressor);
  free(all_buffer);
  free(temp);
  hipFree(&device_input_buffer);
  hipFree(&device_param_refs);
  hipFree(&device_output_buffer);
  (void)fclose(input_file);
  (void)fclose(output_file);

  return 0;
}

int main() {
  return transform_image(INPUT_IMAGE_FILENAME, OUTPUT_IMAGE_FILENAME);
}
